#include"cuda.cuh"
#include"util.h"
void CudaProp()
{
    int device_count;
    hipGetDeviceCount(&device_count);
    for (int i=0; i<device_count; ++i ){
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp,i);
        cout<<"使用GPU"<<i<<":"<<devProp.name<<endl;
        cout<<"SM的数量"<<devProp.multiProcessorCount<<endl;

        cout<< "每个线程块的共享内存大小" << devProp.sharedMemPerBlock / 1024.0  <<"KB" <<endl;
        cout<< "每个线程块的最大线程数"<< devProp.maxThreadsPerBlock<<endl;
        cout<<"设备上一个线程块可用的32位寄存器的数量"<< devProp.regsPerBlock <<endl;
        cout<<"每个EM的最大线程数"<< devProp.maxThreadsPerMultiProcessor<<endl;


        cout<< "设备上多处理器的数量" << devProp.multiProcessorCount<<endl;
        cout<< "设备上EM的最大线程束数" << endl;

        cout<<"=================================================================="<<endl;
    }
}

__device__ float Relu(float p){
    if (p > 0){
        return p;
    }else{
        return 0;
    }
}

