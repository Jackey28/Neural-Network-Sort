#include "hip/hip_runtime.h"
#include"cuda.cuh"
#include"util.h"
void model(float *input,KeysLogits*keylogists)
{
    float *B = (float *)malloc(sizeof(float) * dense1);
    float *W2=(float *)malloc(sizeof(float) * dense2);
    float *C = (float *)malloc(sizeof(float) * data_size * 4);
    float *bias1=(float*)malloc(sizeof(float)*dense1);
    float *bias2=(float*)malloc(sizeof(float)*1);
    *(B) = 1;
    *(B+1) = 0;
    *(B+2) = 0;
    *(B+3) = 1;

    *(W2) = 1;
    *(W2+1) = 0;
    *(W2+2) = 0;
    *(W2+3) = 1;

    *(bias1) = 1;
    *(bias1+1) = 1;
    *(bias1+2) = 1;
    *(bias1+3) = 1;

    *bias2=1;
    steady_clock::time_point t1=steady_clock::now();


    float *A_d, *B_d,*W2_d,*C_d,*bias1_d,*bias2_d;

    hipMalloc((void **)&A_d, sizeof(float) * data_size);
    hipMalloc((void **)&B_d, sizeof(float) * dense1);
    hipMalloc((void **)&C_d, sizeof(float) * data_size * 4);
    hipMalloc((void **)&W2_d, sizeof(float) * dense2);
    hipMalloc((void **)&bias1_d, sizeof(float) * dense1);
    hipMalloc((void **)&bias2_d, sizeof(float) * 1);

    hipMemcpy( A_d, input, sizeof(float) * data_size, hipMemcpyHostToDevice);
    hipMemcpy( B_d, B, sizeof(float) * dense1, hipMemcpyHostToDevice);
    hipMemcpy( W2_d, W2, sizeof(float) * dense2, hipMemcpyHostToDevice);
    hipMemcpy( bias1_d, bias1, sizeof(float) * dense1, hipMemcpyHostToDevice);
    hipMemcpy( bias2_d, bias2, sizeof(float) * 1, hipMemcpyHostToDevice);


    dim3 grindSize(block_x,block_y);
    dim3 blockSize(threads_number_perblock);

    int total_threads = grindSize.x*grindSize.y*blockSize.x*blockSize.y;
    cout<<"total_threads:"<<total_threads<<endl;
    //int sectionSize = data_size/total_threads;
    

   


    dense_1_4 <<<grindSize, blockSize>>> (A_d, B_d,bias1_d, C_d);
     float *temp = (float *)malloc(sizeof(float) * data_size * 4);
     hipMemcpy( temp, C_d, sizeof(float) * data_size * 4, hipMemcpyDeviceToHost);
     cout<<"********************vector A********************";
     cout<<endl;
     for(int i=0;i<data_size;i++)
     {
         cout<<input[i]<<"    ";
     }
     cout<<endl;
     cout<<"********************matrixc C********************";
     cout<<endl;
     for(int i=0;i<data_size*4;i++)
     {
      cout<<*(temp+i)<<"    ";
      if((i+1)%4==0)
         cout<<endl;
     }
     cout<<endl;
    dense_4_1<<<grindSize, blockSize>>>(C_d,W2_d,bias2_d,A_d);
    steady_clock::time_point t2=steady_clock::now();
    hipMemcpy( keylogists->logits, A_d, sizeof(float) * data_size, hipMemcpyDeviceToHost);



    cout<<"********************vector B********************";
    cout<<endl;
    for(int i=0;i<dense1;i++)
    {
     cout<<*(B+i)<<' ';
    }
    cout<<endl;
    cout<<"********************vector logist********************";
    cout<<endl;
    for(int i=0;i<data_size;i++)
    {
     cout<<*(keylogists->logits+i)<<"    ";
    }
    cout<<endl;

    free(B);
    free(C);
    free(W2);
    free(bias1);
    free(bias2);
    free(temp);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipFree(bias1_d);
    hipFree(W2_d);
    hipFree(bias2_d);
    cout << "block_cover:" << block_cover<<endl;
    cout <<"data_size:"<<data_size<<endl;
    cout <<"total_threads"<<total_threads<<endl;
    duration<double,std::milli> *timeSpan = new duration<double,std::milli>(t2-t1);
    cout<<"consumming of alg:"<<timeSpan->count()<<" ms"<<endl;

}

void CudaProp()
{
    int device_count;
    hipGetDeviceCount(&device_count);
    for (int i=0; i<device_count; ++i ){
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp,i);
        cout<<"使用GPU"<<i<<":"<<devProp.name<<endl;
        cout<<"SM的数量"<<devProp.multiProcessorCount<<endl;

        cout<< "每个线程块的共享内存大小" << devProp.sharedMemPerBlock / 1024.0  <<"KB" <<endl;
        cout<< "每个线程块的最大线程数"<< devProp.maxThreadsPerBlock<<endl;
        cout<<"设备上一个线程块可用的32位寄存器的数量"<< devProp.regsPerBlock <<endl;
        cout<<"每个EM的最大线程数"<< devProp.maxThreadsPerMultiProcessor<<endl;


        cout<< "设备上多处理器的数量" << devProp.multiProcessorCount<<endl;
        cout<< "设备上EM的最大线程束数" << endl;

        cout<<"=================================================================="<<endl;
    }
}
__device__ float relu_(float);
struct Output{
    int thread_id;
    int block_id_x;
    int block_id_y;
    float A;
    float B;
    float result;
};
__device__ int getGlobalIdx_2D_1D(){
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (threads_number_perblock) + threadIdx.y * blockDim.x + threadIdx.x;
    return thread_id;
}
__global__ void dense_1_4(float *M, float *N,float*bias, float *P){
    int thread_id = getGlobalIdx_2D_1D();
    int start_point_thread = thread_id*thread_cover;
    int end_point_thread = (thread_id+1)*thread_cover;
    for (int i = start_point_thread; i< end_point_thread; ++i){
         for(int j=0;j<4;j++)
         {
            *(P + i*4+j)= relu_(M[i] * N[j]+bias[j]);
         }
    }

}
__global__ void dense_4_1(float *M,float*N,float*bias,float*P)
{
    int thread_id=getGlobalIdx_2D_1D();  
    int start_point_thread = thread_id*thread_cover;
    int end_point_thread = (thread_id+1)*thread_cover;
    for(int i=start_point_thread;i<end_point_thread;++i)
    {
        float temp=0;
        for(int j=0;j<4;++j)
        {
            temp+=M[i*4+j]*N[j];

        }
        *(P+i)=temp+(*bias);
    }
}

__device__ float relu_(float p){
    if (p > 0){
        return p;
    }else{
        return 0;
    }
}

