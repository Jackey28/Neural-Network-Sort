#include "hip/hip_runtime.h"
#include"cuda.cuh"
#include"util.h"
void CudaProp()
{
    int device_count;
    hipGetDeviceCount(&device_count);
    for (int i=0; i<device_count; ++i ){
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp,i);
        cout<<"使用GPU"<<i<<":"<<devProp.name<<endl;
        cout<<"SM的数量"<<devProp.multiProcessorCount<<endl;

        cout<< "每个线程块的共享内存大小" << devProp.sharedMemPerBlock / 1024.0  <<"KB" <<endl;
        cout<< "每个线程块的最大线程数"<< devProp.maxThreadsPerBlock<<endl;
        cout<<"设备上一个线程块可用的32位寄存器的数量"<< devProp.regsPerBlock <<endl;
        cout<<"每个EM的最大线程数"<< devProp.maxThreadsPerMultiProcessor<<endl;


        cout<< "设备上多处理器的数量" << devProp.multiProcessorCount<<endl;
        cout<< "设备上EM的最大线程束数" << endl;

        cout<<"=================================================================="<<endl;
    }
}

__device__ float Relu(float p){
    if (p > 0){
        return p;
    }else{
        return 0;
    }
}
__device__ int GetThreadX(){
    return blockIdx.x*blockDim.x+threadIdx.x;
}
__device__ int GetThreadY()
{
    return blockIdx.y*blockDim.y+threadIdx.y;
}
__global__ void Dense2D2D( float * input_d, float *matrix_d,Dimension*dim,float* output_d){
    //[m*n][m*k]
    int thread_cover=dim->d1/(gridDim.x*gridDim.y*blockDim.x);

    int blockId = gridDim.x * blockIdx.y+ blockIdx.x;
    int threadId = blockId * blockDim.x+ threadIdx.x;
    int start_point = threadId*thread_cover;
    int end_point = (threadId+1)*thread_cover;
    // (debugOutput+threadId)->start_point = start_point;
    // (debugOutput+threadId)->end_point = end_point;
    // (debugOutput+threadId)->threadId_x = threadId;


    // *(output_d)=*(matrix_d+10);
    for (int i = start_point; i< end_point; ++i){
        for (int j = 0; j < dim->d3; ++j){
            for (int k = 0; k < dim->d2; ++k){
                *(output_d+i*dim->d3+j) += *(input_d+i*dim->d2+k) * (*(matrix_d+j*dim->d2+k));
                // *(output_d+i*dim->d3+j)=66;

            }
        }
    }

}
__global__ void Dense_1_8(float*input,float*params,float*bias,Dimension*dim,float*output)
{

    int x=GetThreadX();
    int y=GetThreadY();
    while(x<dim->d1)
    {
        while(y<dim->d3)
        {
            *(output+x*(dim->d3)+y)=*(input+x)*(*(params+y))+(*(bias+y));
            y+=blockDim.y*gridDim.y;
        }
        y=GetThreadY();
        x+=blockDim.x*gridDim.x;
    }
}
__global__ void Dense_4_1(float*input,float*params,float*bias,Dimension*dim,float*output)
{
    int x=GetThreadX();
    int y=GetThreadY();
    while(x<dim->d1)
    {
        while(y<dim->d2)
        {
            *(input+x*(dim->d2)+y)*=(*(params+y));
            y+=blockDim.y*gridDim.y;
        }
        y=GetThreadY();
        x+=blockDim.x*gridDim.x;
    }
    x=GetThreadX();
    y=GetThreadY();
    int i=(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
    while(i<dim->d1)
    {
        for(int j=0;j<dim->d2;j++)
            *(output+i)+=*(input+i*dim->d2+j);
        i+=gridDim.x*gridDim.y*blockDim.x*blockDim.y;
    }
    // int i=dim->d2/2;
    // while(x<dim->d1)
    // {
    //     while(i!=0)
    //     {
    //         while(y<i)
    //         {

    //             *(input+x*(dim->d2)+y)+=*(input+x*(dim->d2)+y+i);
    //             y+=blockDim.y*gridDim.y;
    //             __syncthreads();
    //         }
    //         i/=2;
    //     }
    //     y=GetThreadY();
    //     *(output+x)=*(input+x*(dim->d2))+(*(bias+y));
    //     x+=blockDim.x*gridDim.x;
    // }
}
__global__ void max_2D_1D (int dataSize, float * inputD, float * outputD){
    int blockId = gridDim.x * blockIdx.y+ blockIdx.x;
    int totalThreads = gridDim.x*gridDim.y * blockDim.x;
    int threadId = blockId * blockDim.x+ threadIdx.x;
    int threadCover = dataSize/(totalThreads);
    
    int startPoint = threadId*threadCover;
    int endPoint = (threadId+1)*threadCover;
    

    float tmpMax = 0;
    for (int i = startPoint; i < endPoint; ++i ){
        if (inputD[i] < tmpMax){
            continue;
        }else{
            tmpMax = inputD[i];
        }
    }
    *(outputD+threadId) = tmpMax;
}
int max_1D(float * input, int dataSize){
    // int max = 0;
    

    int gridDim_x = 2;
    int gridDim_y = 2;
    int blockDim_x = 2;
    int totalThreads = gridDim_x*gridDim_y*blockDim_x;


    dim3 gridSizeTmp(gridDim_x,gridDim_y);
    dim3 blockSizeTmp(blockDim_x);


    float *maxOutput_d, *maxOutput;
    hipMalloc((void**)&maxOutput_d,sizeof(float)*totalThreads);
    maxOutput = (float*)malloc(sizeof(float)*totalThreads);

    // DebugOutput * debugOutput, *debugOutput_d;
    // debugOutput = (DebugOutput *)malloc(sizeof(DebugOutput)*totalThreads);
    // hipMalloc((void**)&debugOutput_d,sizeof(DebugOutput)*totalThreads);
    
    max_2D_1D<<<gridSizeTmp, blockSizeTmp>>>(dataSize, input, maxOutput_d);
    hipMemcpy(maxOutput, maxOutput_d,sizeof(float)*totalThreads,hipMemcpyDeviceToHost);
    float tmpMax = 0;
    for (int i = 0; i< totalThreads; ++i){
        // cout<<"thread_i_max:"<<*(maxOutput+i)<<endl;
        if (*(maxOutput+i) <= tmpMax){
            continue;
        }else{

            tmpMax = *(maxOutput+i);
        }
    }


    cout<<"max:"<<tmpMax<<endl;
    return tmpMax;

}

