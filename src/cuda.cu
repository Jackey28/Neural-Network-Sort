#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"util.h"
void CudaProp()
{
    int device_count;
    hipGetDeviceCount(&device_count);
    for (int i=0; i<device_count; ++i ){
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp,i);
        cout<<"使用GPU"<<i<<":"<<devProp.name<<endl;
        cout<<"SM的数量"<<devProp.multiProcessorCount<<endl;

        cout<< "每个线程块的共享内存大小" << devProp.sharedMemPerBlock / 1024.0  <<"KB" <<endl;
        cout<< "每个线程块的最大线程数"<< devProp.maxThreadsPerBlock<<endl;
        cout<<"设备上一个线程块可用的32位寄存器的数量"<< devProp.regsPerBlock <<endl;
        cout<<"每个EM的最大线程数"<< devProp.maxThreadsPerMultiProcessor<<endl;


        cout<< "设备上多处理器的数量" << devProp.multiProcessorCount<<endl;
        cout<< "设备上EM的最大线程束数" << endl;

        cout<<"=================================================================="<<endl;
    }
}

__device__ float Relu(float p){
    if (p > 0){
        return p;
    }else{
        return 0;
    }
}
__device__ int GetThreadX(){
    return blockIdx.x*blockDim.x+threadIdx.x;
}
__device__ int GetThreadY()
{
    return blockIdx.y*blockDim.y+threadIdx.y;
}
__global__ void Dense2D2D( float * input_d, float *matrix_d,Dimension*dim,float* output_d){
    //[m*n][m*k]
    int thread_cover=dim->d1/(gridDim.x*gridDim.y*blockDim.x);

    int blockId = gridDim.x * blockIdx.y+ blockIdx.x;
    int threadId = blockId * blockDim.x+ threadIdx.x;
    int start_point = threadId*thread_cover;
    int end_point = (threadId+1)*thread_cover;
    // (debugOutput+threadId)->start_point = start_point;
    // (debugOutput+threadId)->end_point = end_point;
    // (debugOutput+threadId)->threadId_x = threadId;


    // *(output_d)=*(matrix_d+10);
    for (int i = start_point; i< end_point; ++i){
        for (int j = 0; j < dim->d3; ++j){
            for (int k = 0; k < dim->d2; ++k){
                *(output_d+i*dim->d3+j) += *(input_d+i*dim->d2+k) * (*(matrix_d+j*dim->d2+k));
                // *(output_d+i*dim->d3+j)=66;

            }
        }
    }

}
__global__ void Dense_1_8(float*input,float*params,float*bias,Dimension*dim,float*output)
{

    int x=GetThreadX();
    int y=GetThreadY();
    while(x<dim->d1)
    {
        while(y<dim->d3)
        {
            *(output+x*(dim->d3)+y)=*(input+x)*(*(params+y))+(*(bias+y));
            y+=blockDim.y*gridDim.y;
        }
        y=GetThreadY();
        x+=blockDim.x*gridDim.x;
    }
}
__global__ void Dense_4_1(float*input,float*params,float*bias,Dimension*dim,float*output)
{
    int x=GetThreadX();
    int y=GetThreadY();
    while(x<dim->d1)
    {
        while(y<dim->d2)
        {
            *(input+x*(dim->d2)+y)*=(*(params+y));
            y+=blockDim.y*gridDim.y;
        }
        y=GetThreadY();
        x+=blockDim.x*gridDim.x;
    }
    x=GetThreadX();
    y=GetThreadY();
    int i=(blockIdx.x+blockIdx.y*gridDim.x)*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
    while(i<dim->d1)
    {
        for(int j=0;j<dim->d2;j++)
            *(output+i)+=*(input+i*dim->d2+j);
        i+=gridDim.x*gridDim.y*blockDim.x*blockDim.y;
    }
    // int i=dim->d2/2;
    // while(x<dim->d1)
    // {
    //     while(i!=0)
    //     {
    //         while(y<i)
    //         {

    //             *(input+x*(dim->d2)+y)+=*(input+x*(dim->d2)+y+i);
    //             y+=blockDim.y*gridDim.y;
    //             __syncthreads();
    //         }
    //         i/=2;
    //     }
    //     y=GetThreadY();
    //     *(output+x)=*(input+x*(dim->d2))+(*(bias+y));
    //     x+=blockDim.x*gridDim.x;
    // }
}
__global__ void max_2D_1D (int dataSize, float * inputD, float * outputD){
    int blockId = gridDim.x * blockIdx.y+ blockIdx.x;
    int totalThreads = gridDim.x*gridDim.y * blockDim.x;
    int threadId = blockId * blockDim.x+ threadIdx.x;
    int threadCover = dataSize/(totalThreads);
    
    int startPoint = threadId*threadCover;
    int endPoint = (threadId+1)*threadCover;
    

    float tmpMax = 0;
    for (int i = startPoint; i < endPoint; ++i ){
        if (inputD[i] < tmpMax){
            continue;
        }else{
            tmpMax = inputD[i];
        }
    }
    *(outputD+threadId) = tmpMax;
}
float max_1D(float * input, int dataSize){
    // int max = 0;
    

    int gridDim_x = 2;
    int gridDim_y = 2;
    int blockDim_x = 2;
    int totalThreads = gridDim_x*gridDim_y*blockDim_x;


    dim3 gridSizeTmp(gridDim_x,gridDim_y);
    dim3 blockSizeTmp(blockDim_x);


    float *maxOutput_d, *maxOutput;
    hipMalloc((void**)&maxOutput_d,sizeof(float)*totalThreads);
    maxOutput = (float*)malloc(sizeof(float)*totalThreads);

    // DebugOutput * debugOutput, *debugOutput_d;
    // debugOutput = (DebugOutput *)malloc(sizeof(DebugOutput)*totalThreads);
    // hipMalloc((void**)&debugOutput_d,sizeof(DebugOutput)*totalThreads);
    
    max_2D_1D<<<gridSizeTmp, blockSizeTmp>>>(dataSize, input, maxOutput_d);
    hipMemcpy(maxOutput, maxOutput_d,sizeof(float)*totalThreads,hipMemcpyDeviceToHost);
    float tmpMax = 0;
    for (int i = 0; i< totalThreads; ++i){
        // cout<<"thread_i_max:"<<*(maxOutput+i)<<endl;
        if (*(maxOutput+i) <= tmpMax){
            continue;
        }else{

            tmpMax = *(maxOutput+i);
        }
    }


    cout<<"max:"<<tmpMax<<endl;
    return tmpMax;

}

float model(KeysLogits*keysLogits,int dataSize,float*rawData){
    int paramsSize=8;
    float*input=rawData;
    float*params=(float*)malloc(sizeof(float)*paramsSize);
    float*bias=(float*)malloc(sizeof(float)*paramsSize);
    float*output=(float*)malloc(sizeof(float)*dataSize*paramsSize);
    Dimension *dim=(Dimension*)malloc(sizeof(Dimension));
    dim->d1=dataSize;
    dim->d2=1;
    dim->d3=paramsSize;                                                                                                                                                                                                         ;
    for(int i=0;i<paramsSize;i++)
    {
        *(params+i)=2;
        *(bias+i)=0.1;
    }
    cout<<"***********************input***********************"<<endl;
    for(int i=0;i<dataSize;i++)
    {
        cout<<*(input+i)<<' ';
        
    }
    cout<<endl;
    cout<<"***********************params***********************"<<endl;
    for(int i=0;i<paramsSize;i++)
    {
        cout<<*(params+i)<<' ';
    }
    cout<<endl;
    cout<<"***********************bias***********************"<<endl;
    for(int i=0;i<paramsSize;i++)
    {
        cout<<*(bias+i)<<' ';
    }
    cout<<endl;
    float *inputD,*paramsD,*biasD,*outputD;
    Dimension*dimD;
    hipMalloc((void **)&inputD, sizeof(float) * dataSize);
    hipMalloc((void **)&paramsD, sizeof(float) * paramsSize);
    hipMalloc((void **)&biasD, sizeof(float) * paramsSize);
    hipMalloc((void **)&outputD, sizeof(float) * dataSize*paramsSize);
    hipMalloc((void**)&dimD,sizeof(Dimension));

    hipMemcpy( inputD, input, sizeof(float) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy( paramsD, params, sizeof(float) * paramsSize, hipMemcpyHostToDevice);
    hipMemcpy( biasD, bias, sizeof(float) * paramsSize, hipMemcpyHostToDevice);
    hipMemcpy(dimD,dim,sizeof(Dimension),hipMemcpyHostToDevice);
    dim3 grid(4,2);
    dim3 block(2,2);
    Dense_1_8<<<1,block>>>(inputD,paramsD,biasD,dimD,outputD);

    hipMemcpy( output, outputD, sizeof(float) * dataSize*paramsSize, hipMemcpyDeviceToHost);

    cout<<"***********************result***********************"<<endl;
    for(int i=0;i<dataSize*paramsSize;i++)
    {
        cout<<*(output+i)<<"      ";
        if((i+1)%paramsSize==0)
            cout<<endl;
    }



    Dimension *dim2=(Dimension*)malloc(sizeof(Dimension));
    dim2->d1=dataSize;
    dim2->d2=8;
    dim2->d3=4;

    float*params2=(float*)malloc(sizeof(float)*dim2->d2*dim2->d3);

    initializeMatrix(params2);
    float*params2D;
    Dimension *dimD2;

    hipMalloc((void **)&dimD2, sizeof(Dimension));
    hipMalloc((void **)&params2D, sizeof(float) * dim2->d2*dim2->d3);
    hipMemcpy(dimD2,dim2,sizeof(Dimension),hipMemcpyHostToDevice);
    hipMemcpy(params2D,params2,sizeof(float)*dim2->d2*dim2->d3,hipMemcpyHostToDevice);
    float*output2=(float*)malloc(sizeof(float)*dim2->d1*dim2->d3);
    float*output2D;

    hipMalloc((void **)&output2D, sizeof(float) * dim2->d1*dim2->d3);
    cout<<"***********************param2***********************"<<endl;
    for(int i=0;i<dim2->d2*dim2->d3;i++)
    {
        cout<<*(params2+i)<<"      ";
        if((i+1)%dim2->d3==0)
            cout<<endl;
    }
    dim3 grid2(2,2);
    dim3 block2(2);
    Dense2D2D<<<grid2,block2 >>>(outputD,params2D,dimD2,output2D);
    hipMemcpy( output2, output2D, sizeof(float) * dim2->d1*dim2->d3, hipMemcpyDeviceToHost);
    cout<<"***********************result2***********************"<<endl;
    for(int i=0;i<dim2->d1*dim2->d3;i++)
    {
        cout<<*(output2+i)<<"      ";
        if((i+1)%dim2->d3==0)
            cout<<endl;
    }
    
    // Dimension *dim33=(Dimension*)malloc(sizeof(Dimension));
    // Dimension *dim33D;
    dim->d1=dataSize;
    dim->d2=4;
    dim->d3=1;
    // Dimension *dimD2;
    float *final=(float*)malloc(sizeof(float)*dim->d1*dim->d3);

    float *finalD;
    hipMalloc((void **)&finalD, sizeof(float) * dim->d1*dim->d3);
    // hipMalloc((void **)&dimD2, sizeof(Dimension));
    hipMemcpy(dimD2,dim,sizeof(Dimension),hipMemcpyHostToDevice);

    Dense_4_1<<<1,block>>>(output2D,paramsD,biasD,dimD2,finalD);
    hipMemcpy( final, finalD, sizeof(float) * dim->d1*dim->d3, hipMemcpyDeviceToHost);
    keysLogits->logits=final;
    cout<<endl;
    cout<<"***********************result3***********************"<<endl;
    for(int i=0;i<dataSize;i++)
    {
        cout<<*(final+i)<<"      ";
    }
    float max=max_1D(finalD, dim->d1);
    // free(input);
    free(params);
    free(bias);
    free(output);
    free(dim);
    free(params2);
    free(dim2);
    free(output2);
    // free(final);
    hipFree(inputD);
    hipFree(paramsD);
    hipFree(biasD);
    hipFree(outputD);
    hipFree(dimD);
    hipFree(dimD2);
    hipFree(params2D);
    hipFree(output2D);
    hipFree(finalD);


    cout<<"hello world\n";
    return max;
}


