#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"util.h"
#include<iomanip>
#include<iostream>
using namespace std;
#define max(a, b) (a > b ? a : b)
#define relu(a) (a > 0 ? a : 0)
void CudaProp()
{
    int device_count;
    hipGetDeviceCount(&device_count);
    for (int i=0; i<device_count; ++i ){
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp,i);
        cout<<"使用GPU"<<i<<":"<<devProp.name<<endl;
        cout<<"SM的数量"<<devProp.multiProcessorCount<<endl;

        cout<< "每个线程块的共享内存大小" << devProp.sharedMemPerBlock / 1024.0  <<"KB" <<endl;
        cout<< "每个线程块的最大线程数"<< devProp.maxThreadsPerBlock<<endl;
        cout<<"设备上一个线程块可用的32位寄存器的数量"<< devProp.regsPerBlock <<endl;
        cout<<"每个EM的最大线程数"<< devProp.maxThreadsPerMultiProcessor<<endl;


        cout<< "设备上多处理器的数量" << devProp.multiProcessorCount<<endl;
        cout<< "设备上EM的最大线程束数" << endl;

        cout<<"=================================================================="<<endl;
    }
}
__device__ int GetThreadX(){
    return blockIdx.x*blockDim.x+threadIdx.x;
}
__device__ int GetThreadY()
{
    return blockIdx.y*blockDim.y+threadIdx.y;
}

__global__ void Dense_2D_2D(double *AD, double * BD, double * bias, Dimension * dim, double * outputD){
    int tid_x = GetThreadX();//threadIdx.x + blockDim.x*blockIdx.x;
    int tid_y = GetThreadY();//blockIdx.y*blockDim.y+ threadIdx.y;
    while(tid_x < dim->d1){
        while (tid_y < dim->d3){
            double tmpValue = 0;
            for (int i = 0; i< dim->d2;++i){
                tmpValue += *(AD + tid_x *dim->d2 + i) * (*(BD + tid_y +i*dim->d3));
            }
            *(outputD + tid_x*dim->d3 + tid_y) = relu(tmpValue+*(bias+tid_y));
            tmpValue = 0;
            tid_y += blockDim.y * gridDim.y;
        }
        tid_y = blockIdx.y*blockDim.y+ threadIdx.y;
        tid_x += blockDim.x*gridDim.x;
    }
}
__global__ void MAX(double * arrayD, double *outputD, int threadPerBlock, int dataSize){
    __shared__ double cache[512];
    int cacheIndex = threadIdx.x;
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    double tmpMax = 0;
    while(tid < dataSize){
        tmpMax = *(arrayD+tid);//max(*(arrayD+tid), tmpMax);
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = tmpMax;
    
    __syncthreads();
    int i = blockDim.x/2;
    while(i !=0){
        if(cacheIndex < i){
            tmpMax = max( *(cache+cacheIndex),  *(cache+cacheIndex+i));
            cache[cacheIndex] = tmpMax;
        }
        __syncthreads();
            i/=2;
    }
    if(cacheIndex == 0){
        *(outputD+blockIdx.x) = tmpMax;
    }

}
double MAX_1D(double *inputD, int dataSize){
    int threadPerBlock = 512;
    int blocksPerGrid  = (dataSize+threadPerBlock)/threadPerBlock;
    cout<<"blockPerGrid:"<<blocksPerGrid<<endl;
    double *input;
    input = (double*)malloc(sizeof(double)*dataSize);

    hipMemcpy(input, inputD, sizeof(double)*dataSize, hipMemcpyDeviceToHost);

    double *maxOutput_d, *maxOutput;
    hipMalloc((void**)&maxOutput_d,sizeof(double)*blocksPerGrid);
    maxOutput = (double*)malloc(sizeof(double)*blocksPerGrid);

    MAX<<<blocksPerGrid, threadPerBlock>>>(inputD, maxOutput_d, threadPerBlock, dataSize);

    hipMemcpy(maxOutput, maxOutput_d, sizeof(double)*blocksPerGrid, hipMemcpyDeviceToHost);
 //   cout<<"MAX:"<<*(maxOutput)<<endl;
    int maxValue = 0;
    for (int i = 0 ; i < blocksPerGrid; ++i){
        maxValue = max(*(maxOutput+i),maxValue);
    }
   // cout<<"MAX in MAX_1D:"<<maxValue<<endl;
    return maxValue;
}

double model(KeysLogits*keysLogits,int dataSize,double*rawData){
    cout<<setprecision(6);
    int paramsSize=8;

    Dimension *dim=(Dimension*)malloc(sizeof(Dimension));
    dim->d1=dataSize;
    dim->d2=1;
    dim->d3=paramsSize;
    cout<<"dataSize:"<<dim->d1<<endl;;                               

    double * weights_1_8, *weights_8_4, *weights_4_1, *bias_1_8, *bias_8_4, *bias_4_1;
    double*input=rawData;
    double*output=(double*)malloc(sizeof(double)*dataSize*paramsSize);
	weights_1_8 = (double*)malloc(sizeof(double)*1*8);
	weights_8_4 = (double*)malloc(sizeof(double)*8*4);
	weights_4_1 = (double*)malloc(sizeof(double)*4*1);
	bias_1_8 = (double*)malloc(sizeof(double)*8);
	bias_8_4 = (double*)malloc(sizeof(double)*4);
    bias_4_1 = (double*)malloc(sizeof(double)*1);


    cout<<setprecision(8); 
    initializeWeightsAndBias(weights_1_8, bias_1_8, weights_8_4, bias_8_4, weights_4_1, bias_4_1);
    double *inputD,*paramsD,*biasD,*outputD;
    Dimension*dimD;
    hipMalloc((void **)&inputD, sizeof(double) * dataSize);
    hipMalloc((void **)&paramsD, sizeof(double) * paramsSize);
    hipMalloc((void **)&biasD, sizeof(double) * paramsSize);
    hipMalloc((void **)&outputD, sizeof(double) * dataSize*paramsSize);
    hipMalloc((void**)&dimD,sizeof(Dimension));

    hipMemcpy( inputD, input, sizeof(double) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy( paramsD, weights_1_8, sizeof(double) * paramsSize, hipMemcpyHostToDevice);
    hipMemcpy( biasD, bias_1_8, sizeof(double) * paramsSize, hipMemcpyHostToDevice);
    hipMemcpy(dimD,dim,sizeof(Dimension),hipMemcpyHostToDevice);
    dim3 grid(16384,8);
    dim3 block(1024,1);

    steady_clock::time_point Start = steady_clock::now();
    Dense_2D_2D<<<grid,block>>>(inputD,paramsD,biasD,dimD,outputD);
    
    Dimension *dim2=(Dimension*)malloc(sizeof(Dimension));
    dim2->d1=dataSize;
    dim2->d2=8;
    dim2->d3=4;

    double*params2D;
    double*bias2D;
    Dimension *dimD2;
    hipMalloc((void **)&dimD2, sizeof(Dimension));
    hipMalloc((void **)&params2D, sizeof(double) * dim2->d2*dim2->d3);
    hipMalloc((void **)&bias2D, sizeof(double) * dim2->d3);
    hipMemcpy(dimD2,dim2,sizeof(Dimension),hipMemcpyHostToDevice);
    hipMemcpy(params2D,weights_8_4,sizeof(double)*dim2->d2*dim2->d3,hipMemcpyHostToDevice);
    hipMemcpy(bias2D,bias_8_4,sizeof(double)*dim2->d3,hipMemcpyHostToDevice);
    double*output2=(double*)malloc(sizeof(double)*dim2->d1*dim2->d3);
    double*output2D;
    hipMalloc((void **)&output2D, sizeof(double) * dim2->d1*dim2->d3);

    Dense_2D_2D<<<grid,block >>>(outputD,params2D,bias2D,dimD2,output2D);
    dim->d1=dataSize;
    dim->d2=4;
    dim->d3=1;
    Dimension *dimD3;
    double *final=(double*)malloc(sizeof(double)*dim->d1*dim->d3);

    double *finalD;
    double*params3D;
    double*bias3D;
    hipMalloc((void **)&finalD, sizeof(double) * dim->d1*dim->d3);
    hipMalloc((void **)&params3D, sizeof(double) * dim->d2);
    hipMalloc((void **)&bias3D, sizeof(double) * dim->d3);
    hipMalloc((void **)&dimD3, sizeof(Dimension));

    hipMemcpy( params3D, weights_4_1, sizeof(double) * dim->d2, hipMemcpyHostToDevice);
    hipMemcpy(dimD3,dim,sizeof(Dimension),hipMemcpyHostToDevice);
    hipMemcpy(bias3D,bias_4_1,sizeof(double)*dim->d3,hipMemcpyHostToDevice);

    Dense_2D_2D<<<grid,block>>>(output2D,params3D,bias3D,dimD3,finalD);
    hipMemcpy(final,finalD,sizeof(double)*dim->d1*dim->d3,hipMemcpyDeviceToHost);
    steady_clock::time_point nonMax = steady_clock::now();
    double max=MAX_1D(finalD, dim->d1);
    
    steady_clock::time_point end = steady_clock::now();
    duration<double, std::milli> *timePredicte = new duration<double, std::milli>(end -Start);
    duration<double, std::milli> *timeNonMax = new duration<double, std::milli>(nonMax -Start);
    cout <<endl<< "consumming of predict:" << timePredicte->count() << " ms" << endl;
    cout <<endl<< "consumming of nonMax:" << timeNonMax->count() << " ms" << endl;

    keysLogits->logits=final;
    // free(input);
    free(output);
    free(dim);
    free(dim2);
    free(output2);
    // free(final);
    hipFree(inputD);
    hipFree(paramsD);
    hipFree(biasD);
    hipFree(outputD);
    hipFree(dimD);
    hipFree(dimD2);
    hipFree(params2D);
    hipFree(output2D);
    hipFree(finalD);


    return max;
}


